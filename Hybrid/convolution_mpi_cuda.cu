#include "hip/hip_runtime.h"
#include <mpi.h>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

using namespace std;
using namespace cv;

// 👉 CUDA clamp helper
__device__ uchar clamp_cuda(float val) {
    return (val < 0.0f) ? 0 : ((val > 255.0f) ? 255 : (uchar)val);
}

// 👉 CUDA kernel
__global__ void cudaConvolution(uchar* input, uchar* output, float* kernel, int rows, int cols, int kSize) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = kSize / 2;

    if (i < rows && j < cols) {
        float sum = 0.0f;
        for (int ki = 0; ki < kSize; ++ki) {
            for (int kj = 0; kj < kSize; ++kj) {
                int ni = i + ki - offset;
                int nj = j + kj - offset;
                if (ni >= 0 && ni < rows && nj >= 0 && nj < cols) {
                    sum += kernel[ki * kSize + kj] * input[ni * cols + nj];
                }
            }
        }
        output[i * cols + j] = clamp_cuda(sum);
    }
}

// 👉 Host helper to run the CUDA kernel with timing
void applyConvolutionCUDA(Mat& input, Mat& output, vector<vector<float>>& kernel, int rank) {
    int rows = input.rows;
    int cols = input.cols;
    int kSize = kernel.size();
    size_t imgSize = rows * cols * sizeof(uchar);
    size_t kernelSize = kSize * kSize * sizeof(float);

    uchar *d_input, *d_output;
    float *d_kernel;

    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_output, imgSize);
    hipMalloc(&d_kernel, kernelSize);

    hipMemcpy(d_input, input.data, imgSize, hipMemcpyHostToDevice);

    float h_kernel[kSize * kSize];
    for (int i = 0; i < kSize; ++i)
        for (int j = 0; j < kSize; ++j)
            h_kernel[i * kSize + j] = kernel[i][j];

    hipMemcpy(d_kernel, h_kernel, kernelSize, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((cols + threads.x - 1) / threads.x, (rows + threads.y - 1) / threads.y);

    // CUDA kernel timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    cudaConvolution<<<blocks, threads>>>(d_input, d_output, d_kernel, rows, cols, kSize);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("✅ Rank %d: CUDA kernel took %.6f seconds\n", rank, milliseconds / 1000.0f);

    hipMemcpy(output.data, d_output, imgSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);
}

int main(int argc, char** argv) {
    MPI_Init(&argc, &argv);
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    double start_time = MPI_Wtime();  // 🕒 Start global timer

    vector<vector<float>> kernel = {
        {0, -1, 0},
        {-1, 5, -1},
        {0, -1, 0}
    };

    Mat inputImage, result;
    int rows, cols;

    if (rank == 0) {
        inputImage = imread("../Images/input.png", IMREAD_GRAYSCALE);
        if (inputImage.empty()) {
            cerr << "❌ Error loading image!" << endl;
            MPI_Abort(MPI_COMM_WORLD, 1);
        }
        rows = inputImage.rows;
        cols = inputImage.cols;
        cout << "✅ Input size: " << rows << " x " << cols << endl;
    }

    MPI_Bcast(&rows, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(&cols, 1, MPI_INT, 0, MPI_COMM_WORLD);

    int chunkSize = rows / size;
    int extra = rows % size;
    int startRow = rank * chunkSize + min(rank, extra);
    int localRows = chunkSize + (rank < extra ? 1 : 0);

    Mat localInput(localRows, cols, CV_8UC1);
    Mat localOutput(localRows, cols, CV_8UC1);

    if (rank == 0) {
        for (int r = 0; r < size; ++r) {
            int sRow = r * chunkSize + min(r, extra);
            int rowsToSend = chunkSize + (r < extra ? 1 : 0);
            if (r == 0) {
                inputImage.rowRange(sRow, sRow + rowsToSend).copyTo(localInput);
            } else {
                MPI_Send(inputImage.ptr(sRow), rowsToSend * cols, MPI_UNSIGNED_CHAR, r, 0, MPI_COMM_WORLD);
            }
        }
    } else {
        MPI_Recv(localInput.ptr(), localRows * cols, MPI_UNSIGNED_CHAR, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }

    applyConvolutionCUDA(localInput, localOutput, kernel, rank);

    if (rank == 0) {
        result = Mat(rows, cols, CV_8UC1);
        localOutput.copyTo(result.rowRange(startRow, startRow + localRows));
        for (int r = 1; r < size; ++r) {
            int sRow = r * chunkSize + min(r, extra);
            int rowsToRecv = chunkSize + (r < extra ? 1 : 0);
            Mat temp(rowsToRecv, cols, CV_8UC1);
            MPI_Recv(temp.ptr(), rowsToRecv * cols, MPI_UNSIGNED_CHAR, r, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
            temp.copyTo(result.rowRange(sRow, sRow + rowsToRecv));
        }
        imwrite("../Results/output_hybrid.png", result);
        cout << "✅ MPI + CUDA Hybrid completed. Output saved: ../Results/output_hybrid.png" << endl;
    } else {
        MPI_Send(localOutput.ptr(), localRows * cols, MPI_UNSIGNED_CHAR, 0, 0, MPI_COMM_WORLD);
    }

    double end_time = MPI_Wtime();  // 🕒 Stop global timer

    if (rank == 0) {
        cout << "✅ Total MPI + CUDA Hybrid time: " << (end_time - start_time) << " seconds." << endl;
    }

    MPI_Finalize();
    return 0;
}
